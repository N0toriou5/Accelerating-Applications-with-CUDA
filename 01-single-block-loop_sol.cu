
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop()
{
  /*
   * This kernel does the work of only 1 iteration
   * of the original for loop. Indication of which
   * "iteration" is being executed by this kernel is
   * still available via `threadIdx.x`.
   */

  printf("This is iteration number %d\n", threadIdx.x);
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */
  loop<<<1, 10>>>();
  hipDeviceSynchronize();
}
