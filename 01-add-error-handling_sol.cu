
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  //Error raised was: Error: invalid configuration argument
  // Need to replace for by eliminating N + stride (leave just N)
  //for (int i = idx; i < N + stride; i += stride)
    
     for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}


bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  // size_t threads_per_block = 2048; WRONG! MaX size should be 1024!!!
  size_t threads_per_block = 1024;  
  size_t number_of_blocks = 32;
    
  hipError_t syncErr, asyncErr;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  
  // include CUDA error handling for kernel doubleElements
  
  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();
  
  hipDeviceSynchronize();
  
  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
